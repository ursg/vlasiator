#include "cuda_build_acc_columns.cuh"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>

struct isZero{
   __host__ __device__ bool operator()(const uint x) {
      return x == 0;
   }
};

__host__ void buildAccColumns(OpenBucketHashtable<vmesh::GlobalID,vmesh::LocalID>& vmesh, uint dimension, hipStream_t stream) {

   size_t numBlocks = vmesh.size();

   thrust::device_vector<vmesh::GlobalID> sortedBlockMappedGID(numBlocks);
   thrust::device_vector<vmesh::LocalID> sortedBlockLID(numBlocks);

   // Sort velocity blocks by the relevant direction
   //thrust::sort_by_key(thrust::cuda::par.on(stream),
   //      sortedBlockMappedGID.begin(), sortedBlockMappedGID.end(),
   //      sortedBlockLID);

   // Tag which blocks are start-of-column
   thrust::device_vector<vmesh::LocalID> columnStartLID(numBlocks);

   // Stream-compact the column start indices
   auto newEnd = thrust::remove_if(thrust::cuda::par.on(stream), 
         columnStartLID.begin(), columnStartLID.end(),
         isZero());

   int numColumns = newEnd - columnStartLID.begin(); 

   // Run along the columns and identify their stop
}
